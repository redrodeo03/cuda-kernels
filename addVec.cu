
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addVec(const float* A, const float* B, float* C, int N) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    for(int i=0; i<N; i++) {
        A[i] = static_cast<float>(i);
        B[i] = static_cast<float>(2*i);
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));
    hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice);

    int blocksize = 256;
    int gridsize = (N+blocksize-1)/blocksize;

    addVec<<<gridsize, blocksize>>>(d_a,d_b,d_c,N);
    hipDeviceSynchronize();
    hipMemcpy(C, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Resultant array C:" << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
