#include <iostream>
#include<hip/hip_runtime.h>

__global__ void MatrixAdd(int *A, int *B, int *C, int N) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if( row<N && col<N){
        int index = row*N + col;
        C[index] = A[index] + B[index];
    }
}

int main() {
    int N=4;
    size_t size = N*N*sizeof(int);

    int h_A[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    int h_B[] = {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1};
    int h_C[N * N];

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(4,4);
    dim3 blocksPerGrid(1,1);

    MatrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "Matrix C (Result of A + B):" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}
